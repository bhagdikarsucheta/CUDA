#include "hip/hip_runtime.h"
#include<math.h>
#include<hip/hip_vector_types.h>
#include<>


#define DIMX 1920
#define DIMY 1080
dim3 grid(DIMX, DIMY);


//sinusodial kernel Rob Faber
//Simple kernel to modify vertex positions in sine wave patter

__global__ void kernel(float4* pos, uchar4 *colorPos, unsigned int width, unsigned int height, float time)
{

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;


	//calculate uv coordinates
	float u = x / (float)width;
	float v = y / (float)height;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	//calculate simple sine wave pattern
	float freq = 4.0f;
	float w = sinf(u*freq + time)*cosf(v*freq + time)*0.5f;

	//write output vertex
	pos[y*width + x] = make_float4(u, w, v, 1.0f);
	colorPos[y*width + x].w = 0;
	colorPos[y*width + x].x = 255.0f * 0.5*(1.0f + sinf(w + x));
	colorPos[y*width + x].y = 255.0f*0.5f*(1.0f + sinf(x)*cosf(y));
	colorPos[y*width+x].z = 255.0f*0.5f*(1.0f + sinf(w + time/10.0 ));

}


//wrapper for the __global__ call sets up the kernel call
void launch_kernel(float4 * pos, uchar4 *colorPos, unsigned int mesh_width, unsigned int mesh_height, float time)
{
	//execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	kernel <<< grid, block >>> (pos, colorPos, mesh_width, mesh_height, time);

}


















