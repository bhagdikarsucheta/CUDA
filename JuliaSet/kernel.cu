#include "hip/hip_runtime.h"
#include<>



#define DIMX 1920
#define DIMY 1080
dim3 grid(DIMX, DIMY);

struct CuComplex {
	float r;
	float i;

	__device__ CuComplex(float a, float b) :r(a), i(b) {}
	__device__ float magnitude2(void) {
		return r * r + i * i;
	}

	__device__ CuComplex operator*(const CuComplex& a)
	{
		return CuComplex(r*a.r - i * a.i, i*a.r + r * a.i);
	}

	__device__ CuComplex operator+(const CuComplex& a)
	{
		return CuComplex(r + a.r, i + a.i);
	}
};


__device__ int julia(int x, int y)
{
	const float scale = 1.5;
	float jx = scale * (float)(DIMX / 2 - x) / (DIMX / 2);
	float jy = scale * (float)(DIMY / 2 - y) / (DIMY / 2);

	CuComplex c(-0.8, 0.154);
	CuComplex a(jx, jy);

	int i = 0;
	for (i = 0; i < 200; i++)
	{
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}
	return 1;
}

__global__ void kernel(unsigned char *ptr)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;

	int juliaValue = julia(x, y);
	ptr[offset * 4 + 0] = 0;
	ptr[offset * 4 + 1] = 0;
	ptr[offset * 4 + 2] = 255 * juliaValue;
	ptr[offset * 4 + 3] = 255;

}

void launch_kernel(unsigned char  *pos)
{
	//execute the kernel
 // execute the kernel
	//dim3 block(8, 8, 1);	//8 * 8 *1 threads
	//dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);	//1024 / 8 = 128	blocks , 1024 /8 = 128 blocks means(128*128 = 16384)blocks
	kernel << <grid, 1 >> > (pos);

	//simple_vbo_kernel << < grid, block >> > (pos, mesh_width, mesh_height, time);
}